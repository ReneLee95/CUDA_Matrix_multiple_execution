#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <iostream>

using namespace std;

__global__ void MatrixMul(int* M, int* N, int* P, int Width) {
	int tx, ty, tid;
	tx = blockDim.x * blockIdx.x + threadIdx.x;
	ty = blockDim.y * blockIdx.y + threadIdx.y;
	
	tid = Width * ty + tx;

	int value = 0;
	int mval = 0;
	int nval = 0;

	for (int i = 0; i < Width; i++) {
		mval = M[Width * tx + i];
		nval = N[i * Width + tx];
		value += mval * nval;
	}

	P[tid] = value;
}

void MatrixMulC(int* M, int* N, int* P, int Width) {
	int col = 0;
	int raw = 0;
	int index = 0;
	int Destindex = 0;

	for (col = 0; col < Width; col++) {
		for (raw = 0; raw < Width; raw++) {
			Destindex = col * Width + raw;
			
			for (index = 0; index < Width; index++) {
				P[Destindex] += M[col * Width + index] * N[index * Width + raw];
			}
		}
	}
}

int main() {
	const int MatrixWidth = 12;
	const int MatrixHeight = 12;
	const int MatrixSize = MatrixWidth * MatrixHeight;
	const int BufferSize = MatrixSize * sizeof(int);

	int* M;
	int* N;
	int* P_cuda;
	int* P_C;

	M = (int*)malloc(BufferSize);
	N = (int*)malloc(BufferSize);
	P_cuda = (int*)malloc(BufferSize);
	P_C = (int*)malloc(BufferSize);

	int i = 0;

	for (i = 0; i < MatrixSize; i++) {
		M[i] = i;
		N[i] = i;
		P_cuda = 0;
		P_C = 0;
	}
	
	int* dev_M;
	int* dev_N;
	int* dev_P;

	hipMalloc((void**)&dev_M, BufferSize);
	hipMalloc((void**)&dev_N, BufferSize);
	hipMalloc((void**)&dev_P, BufferSize);
	
	hipMemcpy(dev_M, M, BufferSize,hipMemcpyHostToDevice);
	hipMemcpy(dev_N, N, BufferSize,hipMemcpyHostToDevice);
	
	dim3 Dg(3, 4, 1);
	dim3 Db(4, 3, 1);
	
	MatrixMul << <Dg, Db >> > (dev_M, dev_N, dev_P, 12);
	hipMemcpy(P_cuda, dev_P, BufferSize,hipMemcpyDeviceToHost);
//	MatrixMulC(M, N, P_C, 12);
	bool ResultFlag = true;
	for (i = 0; i < MatrixSize; i++) {
	//	if (P_cuda[i] != P_C[i]) ResultFlag = false;
	}

	if (ResultFlag == true) printf("MatrixMul Result OK!\n");
	else printf("MatrixMul Result Error\n");
		
	hipFree(dev_M);
	hipFree(dev_N);
	hipFree(dev_P);
		
	free(M);
	free(N);
	free(P_cuda);
	free(P_C);

	return 0;
}