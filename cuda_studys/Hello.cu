#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""

__global__ void kernel(void) { }

int main(void) {
	kernel << <1, 1 >> > ();
	printf("Hello, CUDA\n");

	return 0;
}