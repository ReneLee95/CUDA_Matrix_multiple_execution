#include "hip/hip_runtime.h"
﻿#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <vector>

using namespace std;

vector<int> v;

__global__ void KernelFunction(int a, int b, int c) {
	int sum = a + b + c;  
}

int main() {
	KernelFunction << <6, 6 >> > (1, 2, 3);

	cout << "success" << endl;

	for (int i = 0; i < v.size(); i++) {
		cout << v[i] << endl;
	}

	return 0;
}