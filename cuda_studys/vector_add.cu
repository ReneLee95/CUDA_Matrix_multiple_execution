#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdlib.h>

__global__ void VectorAdd(int* a, int* b, int* c,int size) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	c[tid] = a[tid] + b[tid];
}

int main() {
	const int size = 512 * 65535;
	const int BufferSize = size * sizeof(int);

	int* InputA;
	int* InputB;
	int* Result;

	InputA = (int*)malloc(BufferSize);
	InputB = (int*)malloc(BufferSize);
	Result = (int*)malloc(BufferSize);

	int i = 0;

	for (i = 0; i < size; i++) {
		InputA[i] = i;
		InputB[i] = i;
		Result[i] = 0;
	}

	int* dev_A;
	int* dev_B;
	int* dev_R;

	hipMalloc((void**)&dev_A, size * sizeof(int));
	hipMalloc((void**)&dev_B, size * sizeof(int));
	hipMalloc((void**)&dev_R, size * sizeof(int));

	hipMemcpy(dev_A, InputA, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_B, InputB, size * sizeof(int), hipMemcpyHostToDevice);
	VectorAdd << <65535, 512 >> > (dev_A, dev_B, dev_R, size);

	hipMemcpy(Result, dev_R, size * sizeof(int), hipMemcpyDeviceToHost);
	
	for (i = 0; i < 5; i++) {
		printf("Result[%d] : %d ", i, Result[i]);
			
	}
	std::cout << std::endl;

	for (i = size - 5; i < size; i++) {
		printf("Result[%d] : %d ", i, Result[i]);
	}
	std::cout << std::endl;

	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_R);

	free(InputA);
	free(InputB);
	free(Result);

	return 0;
}